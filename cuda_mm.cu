#include "hip/hip_runtime.h"
/*! 
* \brief Matrix multi with shared memory
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
 
__global__ void MatrixMuiOnDevice(int *M,int *N, int *P, int width)
{      
        int x = threadIdx.x;
        int y = threadIdx.y;
         
        float Pervalue = 0;
         
        for (int i = 0; i < width; i++)
        {      
                float Mdlement = M[y * width + i];
                float Ndlement = N[width * i + x];
                 
                Pervalue += Mdlement * Ndlement;
        }
         
        P[y * width + x] = Pervalue;
}
int main()
{      
        int a[30][30],b[30][30],c[30][30];
        int *M, *N, *P;
        int width = 30;
        int NUM = 900;
        dim3 dimBlock(30,30);
        hipEvent_t start,stop;
        float elapsedTime;
        hipEventCreate(&start);
        hipEventCreate(&stop);
 
 
        hipMalloc((void**)&M, 900*sizeof(int));
        hipMalloc((void**)&N, 900*sizeof(int));
        hipMalloc((void**)&P, 900*sizeof(int));
        for(int i = 0; i < 30; i++)
                for(int j = 0; j < 30; j++)
                {
                        a[i][j] = 2;
                        b[i][j] = 3;
                }
 
        hipMemcpy(M,a,NUM*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(N,b,NUM*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(c,P,NUM*sizeof(int),hipMemcpyDeviceToHost);
        hipEventRecord(start,0);
        MatrixMuiOnDevice<<<1,dimBlock>>>(M,N,P,width);
        hipDeviceSynchronize();
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime,start,stop);
 
        printf("%f\n",elapsedTime);
        for(int i = 0; i < 30; i++)
                for(int j = 0; j < 30; j++)
                {
                        printf("%d \n",c[i][j]);
                }
 
        hipFree(M);
        hipFree(N);
        hipFree(P);
        return 0;
}

